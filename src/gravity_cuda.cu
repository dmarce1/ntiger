#include "hip/hip_runtime.h"
#include <ntiger/gravity.hpp>
#include <ntiger/gravity_cuda.hpp>
#include <ntiger/options.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK( a ) if( a != hipSuccess ) printf( "CUDA error on line %i of %s : %s\n", __LINE__, __FILE__, hipGetErrorString(a))

constexpr int DZ = 1;
constexpr int DY = EWALD_NBIN + 1;
constexpr int DX = (EWALD_NBIN + 1) * (EWALD_NBIN + 1);

real_type *eforce;
real_type *epot;

texture<real_type, NDIM> ftex;
texture<real_type, NDIM> ptex;

void set_cuda_ewald_tables(const ewald_table_t &f, const ewald_table_t &phi) {
	CUDA_CHECK(hipMalloc((void** ) &eforce, sizeof(real) * (EWALD_NBIN + 1) * (EWALD_NBIN + 1) * (EWALD_NBIN + 1)));
	CUDA_CHECK(hipMalloc((void** ) &epot, sizeof(real) * (EWALD_NBIN + 1) * (EWALD_NBIN + 1) * (EWALD_NBIN + 1)));
	CUDA_CHECK(hipMemcpy(eforce, f.data(), sizeof(ewald_table_t), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(epot, phi.data(), sizeof(ewald_table_t), hipMemcpyHostToDevice));

	for (int i = 0; i < 2 * NDIM; i++) {
		ftex.addressMode[i] = hipAddressModeClamp;
		ptex.addressMode[i] = hipAddressModeClamp;

	}
	ftex.filterMode = hipFilterModeLinear;
	ptex.filterMode = hipFilterModeLinear;
	ftex.normalized = false;
	ptex.normalized = false;
	size_t offset = 0;
	constexpr int S3 = (EWALD_NBIN + 1) * (EWALD_NBIN + 1) * (EWALD_NBIN + 1);
	hipBindTexture(&offset, ftex, eforce, sizeof(real_type) * S3);
	hipBindTexture(&offset, ptex, epot, sizeof(real_type) * S3);
}

__global__
void gravity_near_kernel(gravity *g, const vect *x, const vect *y, int xsize, int ysize, real h, real m, bool ewald) {
	int base = blockIdx.x * blockDim.x;
	int i = threadIdx.x + base;
	gravity this_g;
	if (i < xsize) {
		this_g.g = vect(0);
		this_g.phi = 0.0;
		for (int j = 0; j < ysize; j++) {
			vect f;
			real phi;
			const auto dx = x[i] - y[j]; // 3 OP
			if (ewald) {
				auto x0 = dx;
				vect sgn(1.0);
				for (int dim = 0; dim < NDIM; dim++) {
					if (x0[dim] < 0.0) {
						x0[dim] = -x0[dim];                         // 3 * 1 OP
						sgn[dim] *= -1.0;                           // 3 * 1 OP
					}
					if (x0[dim] > 0.5) {
						x0[dim] = 1.0 - x0[dim];                   // 3 * 1 OP
						sgn[dim] *= -1.0;                          // 3 * 1 OP
					}
				}
				const real r = abs(x0);
				for (int dim = 0; dim < NDIM; dim++) {
					f[dim] = 0.0;
				}
				phi = 0.0;
				// Skip ewald
				real fmag = 0.0;
				if (r > 1.0e-3) {
					const real dxbin = 0.5 / EWALD_NBIN;                                   // 1 OP
					general_vect<real_type, NDIM> I;
					for (int dim = 0; dim < NDIM; dim++) {
						I[dim] = (x0[dim] / dxbin).get() + real_type(0.5); 					// 3 * 1 OP
					}
					fmag = tex3D(ftex, I[0], I[1], I[2]);
					f = x0 * (fmag / r);													// 4 OP
					phi = tex3D(ptex, I[0], I[1], I[2]);                                                 // 1 OP
				} else {
					phi = 2.8372975;
				}
				const real r3 = r * r * r;													// 2 OP
				if (r > 0.0) {
					if (r > h) {
						phi = phi - 1.0 / r;													// 2 OP
						f = f - x0 / r3;														// 6 OP
					} else {
						const auto h2 = h * h;
						const auto h3inv = 1.0 / (h * h * h);
						phi = phi - (1.5 * h * h - 0.5 * r * r) / (h * h * h);
						f = f - x0 / (h * h * h);
					}
				}
				for (int dim = 0; dim < NDIM; dim++) {
					f[dim] *= sgn[dim];														// 3 OP
				}
			} else {
				const auto r = abs(dx); // 5 OP
				if (r > 0.0) {
					if (r > h) {            // 1 OP
						const auto rinv = 1.0 / r;            //1 OP
						const auto r3inv = rinv * rinv * rinv;            //2 OP
						f = -dx * r3inv;            //6 OP
						phi = -rinv;                //1 OP
					} else {
						const auto h2 = h * h;
						const auto h3inv = 1.0 / (h * h * h);
						f = -dx * h3inv;
						phi = -(1.5 * h2 - 0.5 * r * r) * h3inv;
					}
				}
			}
			this_g.g = this_g.g + f; // 3 OP
			this_g.phi += phi; // 1 OP
		}
		g[i].g = this_g.g * m; // 1 OP
		g[i].phi = this_g.phi * m; // 1 OP
	}
}

std::vector<gravity> gravity_near_cuda(const std::vector<vect> &x, const std::vector<vect> &y) {
	std::vector<gravity> g(x.size());
	const auto threads_per_block = 256;
	if (x.size() > 0) {
		static const bool ewald = options::get().ewald;
		static const real h = options::get().kernel_size;
		static const real m = 1.0 / options::get().problem_size;
		static thread_local gravity *cg = nullptr;
		static thread_local vect *cx = nullptr;
		static thread_local vect *cy = nullptr;
		static thread_local int xmax = 0;
		static thread_local int ymax = 0;
		if (x.size() > xmax) {
			if (xmax > 0) {
				CUDA_CHECK(hipFree(cg));
				CUDA_CHECK(hipFree(cx));
			}
			CUDA_CHECK(hipMalloc((void** ) &cg, sizeof(gravity) * x.size()));
			CUDA_CHECK(hipMalloc((void** ) &cx, sizeof(vect) * x.size()));
			xmax = x.size();
		}
		if (y.size() > ymax) {
			if (ymax > 0) {
				CUDA_CHECK(hipFree(cy));
			}
			CUDA_CHECK(hipMalloc((void** ) &cy, sizeof(vect) * y.size()));
			ymax = y.size();
		}
		CUDA_CHECK(hipMemcpy(cx, x.data(), x.size() * sizeof(vect), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(cy, y.data(), y.size() * sizeof(vect), hipMemcpyHostToDevice));
		dim3 dimBlock(threads_per_block, 1);
		dim3 dimGrid((x.size() + threads_per_block - 1) / threads_per_block, 1);
gravity_near_kernel<<<dimGrid, dimBlock>>>(cg,cx,cy,x.size(),y.size(),h,m,ewald);
						CUDA_CHECK(hipMemcpy(g.data(), cg, x.size() * sizeof(gravity), hipMemcpyDeviceToHost));
	}
	return g;
}

std::vector<gravity> gravity_far_cuda(const std::vector<vect> &x, const std::vector<source> &y) {

}
