#include "hip/hip_runtime.h"
#include <ntiger/cuda_check.hpp>
#include <ntiger/gravity.hpp>
#include <ntiger/gravity_cuda.hpp>
#include <ntiger/options.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>
void yield_to_hpx();
#include <stack>

#define P 512

hipArray *eforce1 = 0;
hipArray *eforce2 = 0;
hipArray *eforce3 = 0;
hipArray *epot = 0;

texture<real_type, hipTextureType3D> ftex1;
texture<real_type, hipTextureType3D> ftex2;
texture<real_type, hipTextureType3D> ftex3;
texture<real_type, hipTextureType3D> ptex;

void set_cuda_ewald_tables(const std::array<ewald_table_t, NDIM> &f, const ewald_table_t &phi) {
	hipExtent volume = make_hipExtent(EWALD_NBIN + 1, EWALD_NBIN + 1, EWALD_NBIN + 1);
	hipChannelFormatDesc fchan1 = hipCreateChannelDesc<real_type>();
	hipChannelFormatDesc fchan2 = hipCreateChannelDesc<real_type>();
	hipChannelFormatDesc fchan3 = hipCreateChannelDesc<real_type>();
	hipChannelFormatDesc pchan = hipCreateChannelDesc<real_type>();
	CUDA_CHECK(hipMalloc3DArray(&eforce1, &fchan1, volume));
	CUDA_CHECK(hipMalloc3DArray(&eforce2, &fchan2, volume));
	CUDA_CHECK(hipMalloc3DArray(&eforce3, &fchan3, volume));
	CUDA_CHECK(hipMalloc3DArray(&epot, &pchan, volume));

	hipMemcpy3DParms fcopy1 = { 0 };
	fcopy1.srcPtr = make_hipPitchedPtr((void*) f[0].data(), volume.width * sizeof(real_type), volume.width, volume.height);
	fcopy1.dstArray = eforce1;
	fcopy1.extent = volume;
	fcopy1.kind = hipMemcpyHostToDevice;
	CUDA_CHECK(hipMemcpy3D(&fcopy1));

	hipMemcpy3DParms fcopy2 = { 0 };
	fcopy2.srcPtr = make_hipPitchedPtr((void*) f[1].data(), volume.width * sizeof(real_type), volume.width, volume.height);
	fcopy2.dstArray = eforce2;
	fcopy2.extent = volume;
	fcopy2.kind = hipMemcpyHostToDevice;
	CUDA_CHECK(hipMemcpy3D(&fcopy2));

	hipMemcpy3DParms fcopy3 = { 0 };
	fcopy3.srcPtr = make_hipPitchedPtr((void*) f[2].data(), volume.width * sizeof(real_type), volume.width, volume.height);
	fcopy3.dstArray = eforce3;
	fcopy3.extent = volume;
	fcopy3.kind = hipMemcpyHostToDevice;
	CUDA_CHECK(hipMemcpy3D(&fcopy3));

	hipMemcpy3DParms pcopy = { 0 };
	pcopy.srcPtr = make_hipPitchedPtr((void*) phi.data(), volume.width * sizeof(real_type), volume.width, volume.height);
	pcopy.dstArray = epot;
	pcopy.extent = volume;
	pcopy.kind = hipMemcpyHostToDevice;
	CUDA_CHECK(hipMemcpy3D(&pcopy));

	for (int i = 0; i < NDIM; i++) {
		ftex1.addressMode[i] = hipAddressModeClamp;
		ftex2.addressMode[i] = hipAddressModeClamp;
		ftex3.addressMode[i] = hipAddressModeClamp;
		ptex.addressMode[i] = hipAddressModeClamp;

	}
	ftex1.filterMode = hipFilterModeLinear;
	ftex2.filterMode = hipFilterModeLinear;
	ftex3.filterMode = hipFilterModeLinear;
	ptex.filterMode = hipFilterModeLinear;
	ftex1.normalized = false;
	ftex2.normalized = false;
	ftex3.normalized = false;
	ptex.normalized = false;
	CUDA_CHECK(hipBindTextureToArray(ftex1, eforce1, fchan1));
	CUDA_CHECK(hipBindTextureToArray(ftex2, eforce2, fchan2));
	CUDA_CHECK(hipBindTextureToArray(ftex3, eforce3, fchan3));
	CUDA_CHECK(hipBindTextureToArray(ptex, epot, pchan));
}

__global__


                __global__
void direct_gravity_kernel(gravity *__restrict__ g, const vect *x, const source *y, int xsize, int ysize, real h, bool ewald) {
	__shared__ gravity
	this_g[P];
	const real h2 = h * h;
	const real h2t15 = 1.5 * h * h;
	const real h3inv = 1.0 / (h * h * h);
	const int i = blockIdx.x;
	const int l = threadIdx.x;
	this_g[l].g = vect(0);
	this_g[l].phi = 0.0;
#pragma loop unroll 128
	for (int j = l; j < ysize; j += P) {
		vect f;
		real phi;
		const auto dx = x[i] - y[j].x; // 3 OP
		const auto m = y[j].m;
		auto x0 = dx;
		vect sgn(1.0);
		if (ewald) {
#pragma loop unroll 3
			for (int dim = 0; dim < NDIM; dim++) {
				if (x0[dim] < 0.0) {
					x0[dim] = -x0[dim];                         // 3 * 1 OP
					sgn[dim] *= -1.0;                           // 3 * 1 OP
				}
				if (x0[dim] > 0.5) {
					x0[dim] = 1.0 - x0[dim];                   // 3 * 1 OP
					sgn[dim] *= -1.0;                          // 3 * 1 OP
				}
			}
		}
		const real r2 = x0.dot(x0);
#pragma loop unroll 3
		for (int dim = 0; dim < NDIM; dim++) {
			f[dim] = 0.0;
		}
		phi = 0.0;
		if (r2 > 0.0) {
			const auto rinv = rsqrt(r2);            //1 OP
			const auto r3inv = rinv * rinv * rinv;            //2 OP
			if (r2 > h2) {
				phi = phi - rinv;													// 2 OP
				f = f - x0 * r3inv;														// 6 OP
			} else {
				phi = phi - (h2t15 - 0.5 * r2) * h3inv;							//4 OP
				f = f - x0 * h3inv;                                              // 6 OP
			}
			if (ewald) {
#pragma loop unroll 3
				for (int dim = 0; dim < NDIM; dim++) {
					f[dim] *= sgn[dim];														// 3 OP
				}
			}
		}
		this_g[l].g += f * m;                                         // 3 OP
		this_g[l].phi += phi * m;                                      // 1 OP
	}
	__syncthreads();
	for (int N = P / 2; N > 0; N >>= 1) {
		if (l < N) {
			this_g[l].g += this_g[l + N].g;
			this_g[l].phi += this_g[l + N].phi;
		}
		__syncthreads();
	}
	g[i].g = this_g[0].g; // 1 OP
	g[i].phi = this_g[0].phi; // 1 OP
}

__global__
void ewald_gravity_kernel(gravity *__restrict__ g, const vect *x, const source *y, int xsize, int ysize, real h) {

	__shared__ gravity
	this_g[P];
	const real dxbininv = (EWALD_NBIN << 1);                                   // 1 OP
	const int i = blockIdx.x;
	const int l = threadIdx.x;
	this_g[l].g = vect(0);
	this_g[l].phi = 0.0;
#pragma loop unroll 128
	for (int j = l; j < ysize; j += P) {
		vect f;
		real phi;
		const auto dx = x[i] - y[j].x; // 3 OP
		const auto m = y[j].m;
		auto x0 = dx;
		vect sgn(1.0);
#pragma loop unroll 3
		for (int dim = 0; dim < NDIM; dim++) {
			if (x0[dim] < 0.0) {
				x0[dim] = -x0[dim];                         // 3 * 1 OP
				sgn[dim] *= -1.0;                           // 3 * 1 OP
			}
			if (x0[dim] > 0.5) {
				x0[dim] = 1.0 - x0[dim];                   // 3 * 1 OP
				sgn[dim] *= -1.0;                          // 3 * 1 OP
			}
		}
		const real r2 = x0.dot(x0);
#pragma loop unroll 3
		for (int dim = 0; dim < NDIM; dim++) {
			f[dim] = 0.0;
		}
		phi = 0.0;
		if (r2 > 0.0) {
			general_vect<float, NDIM> I;
			for (int dim = 0; dim < NDIM; dim++) {
				I[dim] = (x0[dim] * dxbininv).get() + 0.5; // 3 * 2 OP
			}
			f[0] += tex3D(ftex1, I[0], I[1], I[2]); // 2 OP
			f[1] += tex3D(ftex2, I[0], I[1], I[2]); // 2 OP
			f[2] += tex3D(ftex3, I[0], I[1], I[2]); // 2 OP
			phi += tex3D(ptex, I[0], I[1], I[2]); // 2 OP
			for (int dim = 0; dim < NDIM; dim++) {
				f[dim] *= sgn[dim];														// 3 OP
			}
		}
		this_g[l].g += f * m;                                         // 3 OP
		this_g[l].phi += phi * m;                                      // 1 OP
	}
	__syncthreads();
	for (int N = P / 2; N > 0; N >>= 1) {
		if (l < N) {
			this_g[l].g += this_g[l + N].g;
			this_g[l].phi += this_g[l + N].phi;
		}
		__syncthreads();
	}
	g[i].g = this_g[0].g; // 1 OP
	g[i].phi = this_g[0].phi; // 1 OP

}

struct context {
	gravity *g;
	vect *x;
	source *y;
	std::size_t xsize;
	std::size_t ysize;
	hipStream_t stream;
};

std::stack<context> contexts;
std::atomic<int> lock(0);

context pop_context(std::size_t xs, std::size_t ys) {
	while (lock++ != 0) {
		lock--;
	}
	if (contexts.empty()) {
		context ctx;
		ctx.x = nullptr;
		ctx.y = nullptr;
		ctx.g = nullptr;
		ctx.xsize = 0;
		ctx.ysize = 0;
		hipStreamCreate(&ctx.stream);
		contexts.push(ctx);
	}
	context ctx = contexts.top();
	contexts.pop();
	if (ctx.xsize < xs) {
		printf("X %li\n", xs);
		if (ctx.x != nullptr) {
			CUDA_CHECK(hipFree((void* )ctx.x));
			CUDA_CHECK(hipFree((void* )ctx.g));
		}
		CUDA_CHECK(hipMalloc((void** ) &ctx.g, sizeof(gravity) * xs));
		CUDA_CHECK(hipMalloc((void** ) &ctx.x, sizeof(vect) * xs));
		ctx.xsize = xs;
	}
	if (ctx.ysize < ys) {
		printf("Y %li\n", ys);
		if (ctx.y != nullptr) {
			CUDA_CHECK(hipFree((void* )ctx.y));
		}
		CUDA_CHECK(hipMalloc((void** ) &ctx.y, sizeof(source) * ys));
		ctx.ysize = ys;
	}
	lock--;
	return ctx;
}

void push_context(context ctx) {
	while (lock++ != 0) {
		lock--;
	}
	contexts.push(ctx);
	lock--;
}

pinned_vector<gravity> direct_gravity_cuda(const pinned_vector<vect> &x, const pinned_vector<source> &y) {
//	printf( "<-\n" );
	pinned_vector<gravity> g(x.size());
	double start, stop;

	if (x.size() > 0 && y.size() > 0) {
		bool ewald = options::get().ewald;
		real h = options::get().kernel_size;
		auto ctx = pop_context(x.size(), y.size());
		if (true) {
			start = std::chrono::duration_cast < std::chrono::milliseconds > (std::chrono::system_clock::now().time_since_epoch()).count() / 1000.0;
		}
		CUDA_CHECK(hipMemcpy(ctx.x, x.data(), x.size() * sizeof(vect), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(ctx.y, y.data(), y.size() * sizeof(source), hipMemcpyHostToDevice));

		hipEvent_t event;
		hipEventCreate(&event);

		direct_gravity_kernel<<<x.size(),P,0,ctx.stream>>>(ctx.g,ctx.x,ctx.y,x.size(),y.size(), h, ewald);
		hipEventRecord(event, ctx.stream);

		while (hipEventQuery(event) != hipSuccess) {
			yield_to_hpx();
		}

		CUDA_CHECK(hipMemcpy(g.data(), ctx.g, x.size() * sizeof(gravity), hipMemcpyDeviceToHost));

		push_context(ctx);
		if (true) {
			static double last_display = 0.0;
			static double t = 0.0;
			static double flops = 0.0;
			stop = std::chrono::duration_cast < std::chrono::milliseconds > (std::chrono::system_clock::now().time_since_epoch()).count() / 1000.0;
			t += stop - start;
			flops += x.size() * y.size() * 35;
			//	if (t > 0.0) {
			if (t > last_display + 1.0) {
//				printf("DIRECT %e TFLOPS\n", flops / 1024.0 / 1024.0 / 1024.0 / t / 1024.0);
				last_display = t;
			}

		}

	}
//	printf( ">-\n" );
	return g;
}

pinned_vector<gravity> ewald_gravity_cuda(const pinned_vector<vect> &x, const pinned_vector<source> &y) {
//	printf( "<-\n" );
	pinned_vector<gravity> g(x.size());
	double start, stop;
	if (x.size() > 0 && y.size() > 0) {
		bool ewald = options::get().ewald;
		real h = options::get().kernel_size;
		auto ctx = pop_context(x.size(), y.size());
		if (true) {
			start = std::chrono::duration_cast < std::chrono::milliseconds > (std::chrono::system_clock::now().time_since_epoch()).count() / 1000.0;
		}
		CUDA_CHECK(hipMemcpy(ctx.x, x.data(), x.size() * sizeof(vect), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(ctx.y, y.data(), y.size() * sizeof(source), hipMemcpyHostToDevice));

		hipEvent_t event;
		hipEventCreate(&event);

		ewald_gravity_kernel<<<x.size(), P,0,ctx.stream>>>(ctx.g,ctx.x,ctx.y,x.size(),y.size(), h);
		hipEventRecord(event, ctx.stream);

		while (hipEventQuery(event) != hipSuccess) {
			yield_to_hpx();
		}

		CUDA_CHECK(hipMemcpy(g.data(), ctx.g, x.size() * sizeof(gravity), hipMemcpyDeviceToHost));

		push_context(ctx);

		if (true) {
			static double last_display = 0.0;
			static double t = 0.0;
			static double flops = 0.0;
			stop = std::chrono::duration_cast < std::chrono::milliseconds > (std::chrono::system_clock::now().time_since_epoch()).count() / 1000.0;
			t += stop - start;
			flops += x.size() * y.size() * 100.0;
			if (t > last_display + 1.0) {
				//			printf("%e TFLOPS\n", flops / 1024.0 / 1024.0 / 1024.0 / t / 1024.0);
				last_display = t;
			}

		}
	}
//	printf( ">-\n" );
	return g;
}
